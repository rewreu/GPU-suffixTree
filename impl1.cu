#include "hip/hip_runtime.h"
#include "implementation.h"

__global__ void constructSuffixTree(Node* root, char16_t* text, int* indices, int totalLength, int numStrings){
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	const int nThreads = blockDim.x*gridDim.x;
	const int iter = numStrings%nThreads == 0? numStrings/nThreads : numStrings/nThreads+1;

	for(int i = 0; i < iter; i++){
		int dataid = tid + i*nThreads;
		if(dataid < numStrings){
			int start = indices[dataid];
			int end = dataid == numStrings-1? totalLength : indices[dataid+1];
			for(; text[start] != '#'; start++){
			char16_t c = text[start];
			char16_t index = charToIndex(c);
				Node** address = &(root->children[index]);
				Node* child = *address;
				if(child == NULL){
					child = createNode(start,end);
					addNode(address,child,text);
				} else {
					child = createNode(start,end);
					combineNode(address,child,text);
				}
			}
		}
	}
}

char16_t* impl1(char16_t* text, int* indices, int totalLength, int numStrings, int bsize, int bcount){
	Timer timer;
	Node root;
	root.start=0;
	root.end=0;
	for(int i = 0; i < NUM_CHILDREN; i++)
		root.children[i] = NULL;
	root.suffixIndex = 0;

	char16_t* d_text = NULL;
        int* d_indices = NULL;
	Node* d_root = NULL;

        hipMalloc((void**)&d_text, sizeof(char16_t)*totalLength);
        hipMalloc((void**)&d_indices, sizeof(int)*numStrings);
        hipMalloc((void**)&d_root, sizeof(Node));

        hipMemcpy(d_text, text, sizeof(char16_t)*totalLength, hipMemcpyHostToDevice);
        hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);
	hipMemcpy(d_root,&root,sizeof(Node),hipMemcpyHostToDevice);

	timer.set();

	constructSuffixTree<<<bcount,bsize>>>(d_root,d_text,d_indices,totalLength,numStrings);

	CUDAErrorCheck(hipPeekAtLastError());
	CUDAErrorCheck(hipDeviceSynchronize());
	
	cout << "impl1 stringPerThread running time: " << timer.get() << " ms" << endl;

//	printTree<<<1,1>>>(d_root,d_text);
	//hipDeviceSynchronize();

	char16_t* output = NULL;
	int size = getSerialSuffixTree(d_root,d_text,&output);
	printf("Output size: %d\n",size);

	// free
	hipFree(d_text);
	hipFree(d_indices);
	hipFree(d_root);

	return output;
}

