#include "hip/hip_runtime.h"
#include "implementation.h"

__device__ int getEndOfSuffix(int start, char* text){
	int i = start;
	while(text[i] != '$') i++;
	return i+1;
}

__global__ void constructSuffixTree(Node* root, 
		char* text, int* indices, int* suffixes, 
		int totalLength, int numStrings, int numSuffixes){
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	const int nThreads = blockDim.x*gridDim.x;
	const int iter = numSuffixes%nThreads == 0? numSuffixes/nThreads : numSuffixes/nThreads+1;

	for(int i = 0; i < iter; i++){
		int dataid = tid + i*nThreads;
		if(dataid < numSuffixes){
			int start = suffixes[dataid];
			int end = getEndOfSuffix(start,text);

			char c = text[start];
			char index = charToIndex(c);
			Node** address = &(root->children[index]);
			Node* child = *address;
			if(child == NULL){
				child = createNode(start,end);
				addNode(address,child,text);
			} else {
				child = createNode(start,end);
				combineNode(address,child,text);
			}
		}
	}
}

char* impl2(char* text, int* indices, int* suffixes, 
		int totalLength, int numStrings, int numSuffixes, 
		int bsize, int bcount){
	Timer timer;
	Node root;
	root.start=0;
	root.end=0;
	for(int i = 0; i < NUM_CHILDREN; i++)
		root.children[i] = NULL;
	root.suffixIndex = 0;

	char* d_text = NULL;
        int* d_indices = NULL;
	int* d_suffixes = NULL;
	Node* d_root = NULL;

	hipMalloc((void**)&d_text, sizeof(char)*totalLength);
        hipMalloc((void**)&d_indices, sizeof(int)*numStrings);
	hipMalloc((void**)&d_suffixes,sizeof(int)*numSuffixes);
        hipMalloc((void**)&d_root, sizeof(Node));

        hipMemcpy(d_text, text, sizeof(char)*totalLength, hipMemcpyHostToDevice);
        hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);
        hipMemcpy(d_suffixes, suffixes, sizeof(int)*numSuffixes, hipMemcpyHostToDevice);
	hipMemcpy(d_root,&root,sizeof(Node),hipMemcpyHostToDevice);

	timer.set();

	constructSuffixTree<<<bcount,bsize>>>(d_root,
		d_text,d_indices,d_suffixes,
		totalLength,numStrings,numSuffixes);

	CUDAErrorCheck(hipPeekAtLastError());
	CUDAErrorCheck(hipDeviceSynchronize());

	cout << "impl2 suffixPerThread running time: " << timer.get() << " ms" << endl;
//	printTree<<<1,1>>>(d_root,d_text);
	char* output = NULL;
	int size = getSerialSuffixTree(d_root,d_text,&output);
	printf("Output size: %d\n",size);

	// free
	hipFree(d_text);
	hipFree(d_indices);
	hipFree(d_suffixes);
	hipFree(d_root);

	return output;
}
