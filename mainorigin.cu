#include <iostream>
#include <fstream>
#include <vector>
#include <numeric>
#include "cuda_error_check.h"
#include "implementation.h"
#include "utils.h"
#include "sequential.cpp"
#include <uchar.h>

using namespace std;
int main(int argc, char** argv){
	try {
		//declare and initialize variabls
		string usage =
		"\tCommand line arguments:\n\
                        Input file: E.g., --input in.txt\n\
                        Output path: E.g., --output out.txt\n\
                        Block size: E.g., --bsize 1024\n\
                        Block count: E.g., --bcount 2\n\
                        Method: E.g., --method 1 or 2\n";
		string inputFileName;
		string outputFileName;
		ifstream inputFile;
		ofstream outputFile;
		int bsize = 0, bcount = 0;
		int method = 0;
		int deviceID = 0;
		hipDeviceProp_t deviceProp;
		char* deviceName = NULL;
		int numberOfChildren = 1009; // use a large prime number

		//check that CUDA is supported and get the name of the device
		CUDAErrorCheck(hipSetDevice(deviceID));
		CUDAErrorCheck(hipGetDeviceProperties(&deviceProp, deviceID));
		deviceName = deviceProp.name;
	
		//parse program arguments
		for( int i = 1; i < argc; i++ ){
			if ( !strcmp(argv[i], "--input") && i != argc-1 ) {
				inputFileName = string(argv[i+1]);
				inputFile.open(inputFileName.c_str());
			} else if( !strcmp(argv[i], "--output") && i != argc-1 ) {
				outputFileName = string(argv[i+1]);
				outputFile.open(outputFileName.c_str());
			} else if( !strcmp(argv[i], "--bsize") && i != argc-1 ) {
				bsize = atoi( argv[i+1] );
			} else if( !strcmp(argv[i], "--bcount") && i != argc-1 ) {
				bcount = atoi( argv[i+1] );
			} else if( !strcmp(argv[i], "--method") && i != argc-1 ) {
				method = atoi( argv[i+1] );
			}
		}

		//verify program arguments
		if(!inputFile){
			throw runtime_error("Failed to open specified file: " + inputFileName);
		}
		if(!outputFile){
			throw runtime_error("Failed to open specified file: " + outputFileName);
		}
		if(!inputFile.is_open() || !outputFile.is_open()){
			cerr << "Usage: " << usage;
			throw runtime_error("Initialization error happened: input/output file");
		}
		if(bsize <= 0 || bcount <= 0){
			cerr << "Usage: " << usage;
			throw runtime_error("Initialization error happened: block size/count");
		}
		if(method == 0){
			cerr << "Usage: " << usage;
			throw runtime_error("Initialization error happened: method");
		}

		//parse input file
		vector<string> strings = parseFile(inputFile);
		inputFile.close();
		char* text; //all the strings concatenated into a single string
		int* indices; //the starting index of each string
		int* suffixes; //the starting index of each suffix
		int totalLength; //length of text (includes term sequence)
		int numStrings; //number of strings
		int numSuffixes; //number of suffixes
		//parseStrings(strings,text,indices,suffixes,totalLength,numStrings,numSuffixes);
        preParseStrings(strings,text,indices,
                suffixes,totalLength,numStrings,numSuffixes);
		//print program properties
		cout << "Device: " << deviceName;
		cout << ", bsize: " << bsize << ", bcount: " << bcount;
		cout << ", method: " << method << endl;	
		cout << "Input file: " << inputFileName;
		cout << ", Number of strings: " << numStrings;
		cout << ", Number of suffixes: " << numSuffixes;
		cout << ", total length: " << totalLength << endl;

		// test char16_t
//        char16_t* testChar;
//        testChar = (char16_t*)malloc(2+1);
        char16_t* testChar = (char16_t*)malloc(2+1);
        testChar[0] = 999;
        testChar[1] = 'a';
        cout << ", first test char is: " <<testChar[0] <<", second test char is: " <<testChar[1] << endl;
        int lengthNewText = totalLength/numberOfChildren;
        cout<<"length of text is: "<< length<<endl;


		CUDAErrorCheck(hipDeviceSetLimit(hipLimitMallocHeapSize, 1000000000));
		CUDAErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 50000));
		size_t limit = 0;
		hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
		printf("hipLimitMallocHeapSize: %u\n", (unsigned)limit);
		hipDeviceGetLimit(&limit, hipLimitStackSize);
		printf("hipLimitStackSize: %u\n", (unsigned)limit);

		print_seq_runtime(text);

		char* output = NULL;

		//process method
		switch(method){
		case 1:
			output = impl1(text, 
				indices, 
				totalLength, 
				numStrings, 
				bsize, bcount);
				cout << "text is:"<<text<<"\n";
				cout << "indices 2 is:"<<indices[1]<<"\n";
//				cout << "text is:"<<text;
			break;
		case 2:
			output = impl2(text, 
				indices, 
				suffixes, 
				totalLength, 
				numStrings, 
				numSuffixes,
				bsize, bcount);
			break;
		default:
			cout << "Method " << method << " does not exist. Try method 1 or 2.\n";
			break;
		}

		saveResults(outputFile, output);
		outputFile.close();

		//clean program memory
		free(text);
		free(indices);
		free(suffixes);
		CUDAErrorCheck(hipDeviceReset());

	} catch(const exception& e){
		cerr << e.what() << endl;
		return EXIT_FAILURE;
	} catch(...) {
		cerr << "An exception has occurred." << endl;
		return EXIT_FAILURE;
	}

	return EXIT_SUCCESS;
}
