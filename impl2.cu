#include "hip/hip_runtime.h"
#include "implementation.h"
#include <uchar.h>

__device__ int getEndOfSuffix(int start, char16_t* text){
	int i = start;
	while(!(text[i] == '$' || text[i+2] == '#')) i++;
	return i+1;
}

__global__ void constructSuffixTree(Node* root, 
		char16_t* text, int* indices, int* suffixes,
		int totalLength, int numStrings, int numSuffixes){
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;
	const int nThreads = blockDim.x*gridDim.x;
	const int iter = numSuffixes%nThreads == 0? numSuffixes/nThreads : numSuffixes/nThreads+1;

	for(int i = 0; i < iter; i++){
		int dataid = tid + i*nThreads;
		if(dataid < numSuffixes){
			int start = suffixes[dataid];
			int end = getEndOfSuffix(start,text);

			char16_t c = text[start];
//			char16_t index = charToIndex(c);
			char16_t index = c;
			Node** address = &(root->children[index]);
			Node* child = *address;
			if(child == NULL){
				child = createNode(start,end);
				addNode(address,child,text);
			} else {
				child = createNode(start,end);
				combineNode(address,child,text);
			}
		}
	}
}

char16_t* impl2(char16_t* text, int* indices, int* suffixes,
		int totalLength, int numStrings, int numSuffixes, 
		int bsize, int bcount){
	Timer timer;
	Node root;
	root.start=0;
	root.end=0;
	for(int i = 0; i < NUM_CHILDREN; i++)
		root.children[i] = NULL;
	root.suffixIndex = 0;

	char16_t* d_text = NULL;
        int* d_indices = NULL;
	int* d_suffixes = NULL;
	Node* d_root = NULL;

	hipMalloc((void**)&d_text, sizeof(char16_t)*totalLength);
        hipMalloc((void**)&d_indices, sizeof(int)*numStrings);
	hipMalloc((void**)&d_suffixes,sizeof(int)*numSuffixes);
        hipMalloc((void**)&d_root, sizeof(Node));

        hipMemcpy(d_text, text, sizeof(char16_t)*totalLength, hipMemcpyHostToDevice);
        hipMemcpy(d_indices, indices, sizeof(int)*numStrings, hipMemcpyHostToDevice);
        hipMemcpy(d_suffixes, suffixes, sizeof(int)*numSuffixes, hipMemcpyHostToDevice);
	hipMemcpy(d_root,&root,sizeof(Node),hipMemcpyHostToDevice);

	timer.set();

	constructSuffixTree<<<bcount,bsize>>>(d_root,
		d_text,d_indices,d_suffixes,
		totalLength,numStrings,numSuffixes);

	CUDAErrorCheck(hipPeekAtLastError());
	CUDAErrorCheck(hipDeviceSynchronize());

	cout << "impl2 suffixPerThread running time: " << timer.get() << " ms" << endl;
	printTree<<<1,1>>>(d_root,d_text);
	char16_t* output = NULL;
	int size = getSerialSuffixTree(d_root,d_text,&output);
	printf("Output size: %d\n",size);

	// free
	hipFree(d_text);
	hipFree(d_indices);
	hipFree(d_suffixes);
	hipFree(d_root);

	return output;
}
